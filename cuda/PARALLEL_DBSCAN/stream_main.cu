#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <unistd.h>
#define dimension 15
#define n_points 5000
#define s n_points * n_points

__global__ void distance_matrix_kernel(float *dataset, float *distance_matrix) {
	int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
	if (thread_id < s) {
		int offset = thread_id / n_points;
		int mod = thread_id % n_points;
		float distance = 0;
		for (int j = 0; j < dimension; j++)
			distance += pow(
					dataset[j + offset * dimension]
							- dataset[j + mod * dimension], 2);
		distance_matrix[thread_id] = sqrt(distance);
	}
}

float * CPU_distance_matrix(float *dataset) {
	float *distance_matrix;
	float distance = 0;
	distance_matrix = (float *) malloc(n_points * n_points * sizeof(float));
	for (int i = 0; i < n_points; i++) {
		for (int j = 0; j < n_points; j++) {
			for (int k = 0; k < dimension; k++)
				distance += pow(
						*(dataset + i * dimension + k)
								- *(dataset + j * dimension + k), 2);
			*(distance_matrix + i * n_points + j) = sqrt(distance);
			distance = 0;
		}
	}
	return distance_matrix;
}

int main() {
	float *h_dataset, *h_distance_matrix, *h_shared;
	h_dataset = (float *) malloc(n_points * dimension * sizeof(float));
	h_distance_matrix = (float *) malloc(n_points * n_points * sizeof(float));
	h_shared = (float *) malloc(n_points * n_points * sizeof(float));
	int n = 0;
	double sum_CPU = 0;
	double sum_GPU = 0;
	double sum_shared = 0;
	float *dstmtr;
	FILE *fptr;
	fptr = fopen("/home/federico/cuda-workspace/new_stream/15d_dataset.txt",
			"r");
	while (fscanf(fptr, "%f", &h_dataset[n++]) != EOF)
		;
	fclose(fptr);
	float *d_dataset, *d_distance_matrix, *shared_distance_matrix;
	int dataset_size = n_points * dimension * sizeof(float);
	int distance_matrix_dimension = n_points * n_points * sizeof(float);

	hipMalloc(&d_dataset, dataset_size);
	hipMalloc(&d_distance_matrix, distance_matrix_dimension);
	hipMalloc(&shared_distance_matrix, distance_matrix_dimension);

	hipMemcpy(d_dataset, h_dataset, dataset_size, hipMemcpyHostToDevice);
	hipMemset(d_distance_matrix, 0, distance_matrix_dimension);
	//cudaMemset(shared_distance_matrix, 0, distance_matrix_dimension);

	int gridSize = 24415;
	int blockSize = 1024;
	int nStreams = 257;
	int k = gridSize / nStreams;
	int bytesPerStream = k * blockSize * sizeof(float);
	hipStream_t stream[nStreams];
	for (int i = 0; i < nStreams; ++i) {
		hipStreamCreate(&stream[i]);
	}
	int offset = 0;

	for (int i = 0; i < nStreams; ++i) {
		offset = i * bytesPerStream;
		distance_matrix_kernel<<<95, blockSize, 0, stream[i]>>>(d_dataset,
				d_distance_matrix);
		hipMemcpyAsync(&h_distance_matrix[offset], &d_distance_matrix[offset],
				bytesPerStream, hipMemcpyDeviceToHost, stream[i]);
	}

	for (int i = 0; i < nStreams; ++i) {
		hipStreamDestroy(stream[i]);
	}

	dstmtr = CPU_distance_matrix(h_dataset);
	/*distance_matrix_kernel<<<24415, 1024>>>(d_dataset, d_distance_matrix);
	cudaMemcpy(h_distance_matrix, d_distance_matrix, distance_matrix_dimension,
			cudaMemcpyDeviceToHost);*/
	for (int i = 0; i < n_points * n_points; i++) {
		sum_CPU += *(dstmtr + i);
		sum_GPU += h_distance_matrix[i];
		sum_shared += h_shared[i];
	}
	printf("%f  ---   %f --- %f\n", sum_CPU, sum_GPU, sum_shared);

	free(h_dataset);
	free(h_distance_matrix);
	free(h_shared);
	free(dstmtr);
	hipFree(d_dataset);
	hipFree(d_distance_matrix);
	hipFree(shared_distance_matrix);

	return 0;
}
