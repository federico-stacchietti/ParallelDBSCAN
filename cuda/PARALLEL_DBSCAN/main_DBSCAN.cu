#include <stdio.h>
#include <stdlib.h>
#include "host_DBSCAN.h"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#define dimension 10
#define n_points 30

int main() {
	float *dataset;
	dataset = (float *) malloc(n_points * dimension * sizeof(float));

	int n = 0;
	FILE *fptr;
	fptr = fopen("/home/federico/cuda-workspace/PARALLEL_DBSCAN/10D_random_dataset.txt", "r");
	while (fscanf(fptr, "%f", &dataset[n++]) != EOF);
	fclose(fptr);
	int *ptr;
	ptr = DBSCAN(region_matrix(dataset, 7), 3);
	for (int i = 0; i < n_points; i++) {
		printf("%d\n", *(ptr + i) - 1);
	}


	return 0;
}
